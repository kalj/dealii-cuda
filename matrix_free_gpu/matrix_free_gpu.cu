#include "hip/hip_runtime.h"
/* -*- c-basic-offset:2; tab-width:2; indent-tabs-mode:nil -*-
 *
 * @(#)matrix_free_gpu.cu
 * @author Karl Ljungkvist <karl.ljungkvist@it.uu.se>
 *
 */

#include <deal.II/fe/fe.h>
#include <deal.II/fe/fe_values.h>
#include <deal.II/matrix_free/shape_info.h>
#include <deal.II/grid/filtered_iterator.h>
#include <deal.II/base/graph_coloring.h>

#include "coloring.h"
// #include "hanging_nodes.cuh"
#include "cuda_utils.cuh"


//=============================================================================
// MatrixFreeGpu is an object living on the CPU, but with most of its member
// data residing on the gpu. Here, we keep all the data related to a matrix-free
// evaluation.
//=============================================================================


#define MATRIX_FREE_BKSIZE_CONSTR 128

// helper object for (re)initialization of main class
template <int dim, typename Number>
class ReinitHelper {
private:
  MatrixFreeGpu<dim,Number> *data;

  // host arrays
  std::vector<unsigned int> loc2glob_host;
  std::vector<Point<dim> > quad_points_host;
  std::vector<Number> JxW_host;
  std::vector<Number> inv_jac_host;

  std::vector<unsigned int> constraint_mask_host;

  // local buffers
  std::vector<types::global_dof_index> local_dof_indices;

  FEValues<dim> fe_values;
  // get the translation from default dof numbering to a lexicographic one
  const std::vector<unsigned int> &lexicographic_inv;
  std::vector<unsigned int> lexicographic_dof_indices;

  const unsigned int fe_degree;
  const unsigned int dofs_per_cell;
  const unsigned int qpts_per_cell;

  // TODO: fix update flags
  const UpdateFlags &update_flags;

  // For setting up hanging node constraints
  // HangingNodes<dim> hanging_nodes;

  // for padding
  const unsigned int rowlength;

public:
  ReinitHelper(MatrixFreeGpu<dim,Number>                              *data,
               const Mapping<dim>                                     &mapping,
               const FiniteElement<dim>                               &fe,
               const Quadrature<1>                                    &quad,
               const internal::MatrixFreeFunctions::ShapeInfo<Number> &shape_info,
               const DoFHandler<dim>                                  &dof_handler,
               const UpdateFlags &update_flags)
    : data(data),
      fe_degree(data->fe_degree),
      dofs_per_cell(data->dofs_per_cell),
      qpts_per_cell(data->qpts_per_cell),
      fe_values (mapping, fe, Quadrature<dim>(quad),
                 update_inverse_jacobians | update_quadrature_points |
                 update_values | update_gradients | update_JxW_values),
      lexicographic_inv(shape_info.lexicographic_numbering),
      // hanging_nodes(fe_degree,dof_handler,lexicographic_inv),
      update_flags(update_flags),
      rowlength(data->get_rowlength())
  {
    local_dof_indices.resize(data->dofs_per_cell);
    lexicographic_dof_indices.resize(dofs_per_cell);
  }


  void setup_color_arrays(const unsigned int num_colors);
  void setup_cell_arrays(const unsigned int c);

  template <typename T>
  void get_cell_data(const T& cell,const unsigned int cellid);
  void alloc_and_copy_arrays(const unsigned int c);
};

template <int dim, typename Number>
void ReinitHelper<dim,Number>::setup_color_arrays(const unsigned int num_colors)
{
  data->n_cells.resize(num_colors);
  data->grid_dim.resize(num_colors);
  data->block_dim.resize(num_colors);
  data->loc2glob.resize(num_colors);
  data->constraint_mask.resize(num_colors);

  data->rowstart.resize(num_colors);

  if(update_flags & update_quadrature_points)
    data->quadrature_points.resize(num_colors);

  if(update_flags & update_JxW_values)
    data->JxW.resize(num_colors);

  if(update_flags & update_gradients)
    data->inv_jac.resize(num_colors);
}

template <int dim, typename Number>
void ReinitHelper<dim,Number>::setup_cell_arrays(const unsigned int c)
{
  const unsigned int n_cells = data->n_cells[c];
  const unsigned int cells_per_block = data->cells_per_block;

  // setup kernel parameters
  const unsigned int apply_num_blocks = ceil(n_cells / float(cells_per_block));
  const unsigned int apply_x_num_blocks = round(sqrt(apply_num_blocks)); // get closest to even square.
  const unsigned int apply_y_num_blocks = ceil(double(apply_num_blocks)/apply_x_num_blocks);

  data->grid_dim[c] = dim3(apply_x_num_blocks,apply_y_num_blocks);

  const unsigned int n_dofs_1d = fe_degree+1;

  if(data->parallelization_scheme == MatrixFreeGpu<dim,Number>::scheme_par_in_elem) {

    if(dim==1)
      data->block_dim[c] = dim3(n_dofs_1d*cells_per_block);
    else if(dim==2)
      data->block_dim[c] = dim3(n_dofs_1d*cells_per_block,n_dofs_1d);
    else if(dim==3)
      data->block_dim[c] = dim3(n_dofs_1d*cells_per_block,n_dofs_1d,n_dofs_1d);
  }
  else {
    data->block_dim[c] = dim3(cells_per_block);
  }


  loc2glob_host.resize(n_cells*rowlength);

  if(update_flags & update_quadrature_points)
    quad_points_host.resize(n_cells*rowlength);

  if(update_flags & update_JxW_values)
    JxW_host.resize(n_cells*rowlength);

  if(update_flags & update_gradients)
    inv_jac_host.resize(n_cells*rowlength*dim*dim);

  constraint_mask_host.resize(n_cells);

}

template <int dim, typename Number>
template <typename T>
void ReinitHelper<dim,Number>::get_cell_data(const T& cell, const unsigned int cellid)
{
  cell->get_dof_indices(local_dof_indices);

  for(int i = 0; i < dofs_per_cell; ++i)
    lexicographic_dof_indices[i] = local_dof_indices[lexicographic_inv[i]];

  // setup hanging nodes
  // hanging_nodes.setup_constraints (constraint_mask_host[cellid],
  //                                  lexicographic_dof_indices,
  //                                  cell,cellid);

  memcpy(&loc2glob_host[cellid*rowlength],&lexicographic_dof_indices[0],dofs_per_cell*sizeof(unsigned int));

  fe_values.reinit(cell);

  // quadrature points
  if(update_flags & update_quadrature_points) {
    const std::vector<Point<dim> > & qpts = fe_values.get_quadrature_points();
    memcpy(&quad_points_host[cellid*rowlength],&qpts[0],qpts_per_cell*sizeof(Point<dim>));
  }

  if(update_flags & update_JxW_values) {
    const std::vector<Number > & jxws = fe_values.get_JxW_values();
    memcpy(&JxW_host[cellid*rowlength],&jxws[0],qpts_per_cell*sizeof(Number));
  }

  if(update_flags & update_gradients) {
    const std::vector<DerivativeForm<1,dim,dim> >& jacs = fe_values.get_inverse_jacobians();
    memcpy(&inv_jac_host[cellid*rowlength*dim*dim],&jacs[0],qpts_per_cell*sizeof(DerivativeForm<1,dim,dim>));
  }
}



template <typename T>
void transpose(T *dst, const T *src, const unsigned int N, const unsigned int M)
{
  // src is N X M
  // dst is M X N

  for(int i = 0; i < N; ++i)
    for(int j = 0; j < M; ++j)
      dst[j*N+i] = src[i*M+j];
}

// TODO: if a unified gpuarray / point would exist, only need one template argument
template <typename T>
void transpose_inplace(std::vector<T> &a_host,
                       const unsigned int n, const unsigned int m)
{
  // convert to structure-of-array
  std::vector<T> old(a_host.size());
  old.swap(a_host);

  transpose(&a_host[0],&old[0],n,m);
}


template <typename T1, typename T2>
void alloc_and_copy(T1 **a_dev, std::vector<T2> &a_host,
                    const unsigned int n)
{
  CUDA_CHECK_SUCCESS(hipMalloc(a_dev,n*sizeof(T1)));
  CUDA_CHECK_SUCCESS(hipMemcpy(*a_dev, &a_host[0], n*sizeof(T1),
                                hipMemcpyHostToDevice));
}

template <int dim, typename Number>
void ReinitHelper<dim,Number>::alloc_and_copy_arrays(const unsigned int c)
{
  const unsigned n_cells = data->n_cells[c];

  // local-to-global mapping
  if(data->parallelization_scheme == MatrixFreeGpu<dim,Number>::scheme_par_over_elems) {
    transpose_inplace(loc2glob_host,n_cells, rowlength);
  }
  alloc_and_copy(&data->loc2glob[c],
                 loc2glob_host,
                 n_cells*rowlength);

  // quadrature points
  if(update_flags & update_quadrature_points) {
    if(data->parallelization_scheme == MatrixFreeGpu<dim,Number>::scheme_par_over_elems) {
      transpose_inplace(quad_points_host,n_cells, rowlength);
    }
    alloc_and_copy(&data->quadrature_points[c],
                   quad_points_host,
                   n_cells*rowlength);
  }

  // jacobian determinants/quadrature weights
  if(update_flags & update_JxW_values) {
    if(data->parallelization_scheme == MatrixFreeGpu<dim,Number>::scheme_par_over_elems) {
      transpose_inplace(JxW_host,n_cells, rowlength);
    }
    alloc_and_copy(&data->JxW[c],
                   JxW_host,
                   n_cells*rowlength);
  }
  // inverse jacobians
  if(update_flags & update_gradients) {

    // now this has index order:  cellid*qpts_per_cell*dim*dim + q*dim*dim + i
    // this is not good at all?

    // convert so that all J_11 elements are together, all J_12 elements together, etc.
    // i.e. this index order: i*qpts_per_cell*n_cells + cellid*qpts_per_cell + q
    // this is good for a dof-level parallelization

    transpose_inplace(inv_jac_host,rowlength*n_cells,dim*dim);

    // transpose second time means we get the following index order:
    // q*n_cells*dim*dim + i*n_cells + cellid
    // which is good for an element-level parallelization

    if(data->parallelization_scheme == MatrixFreeGpu<dim,Number>::scheme_par_over_elems) {
      transpose_inplace(inv_jac_host,n_cells*dim*dim, rowlength);
    }
    alloc_and_copy(&data->inv_jac[c], inv_jac_host,
                   n_cells*dim*dim*rowlength);
  }

  alloc_and_copy(&data->constraint_mask[c],constraint_mask_host,n_cells);
}


//=============================================================================
// Initialization function
//=============================================================================

template <int dim, typename Number>
void MatrixFreeGpu<dim,Number>::
reinit(const Mapping<dim>        &mapping,
       const DoFHandler<dim>     &dof_handler,
       const ConstraintMatrix &constraints,
       const Quadrature<1>           &quad,
       const AdditionalData    additional_data)
{

  if(typeid(Number) == typeid(double)) {
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
  }

  use_coloring = additional_data.use_coloring;

  const UpdateFlags &update_flags = additional_data.mapping_update_flags;

  if(additional_data.parallelization_scheme != scheme_par_over_elems &&
     additional_data.parallelization_scheme != scheme_par_in_elem) {
    fprintf(stderr,"Invalid parallelization scheme!\n");
    exit(1);
  }

  this->parallelization_scheme = additional_data.parallelization_scheme;

  free(); // todo, only free if we actually need arrays of different length

  const FiniteElement<dim> &fe = dof_handler.get_fe();

  fe_degree = fe.degree;
  const unsigned int n_dofs_1d = fe_degree+1;
  const unsigned int n_q_points_1d = quad.size();

  Assert(n_dofs_1d == n_q_points_1d,ExcMessage("n_q_points_1d must be equal to fe_degree+1."));

  n_dofs = dof_handler.n_dofs();
  n_cells_tot = dof_handler.get_triangulation().n_active_cells();

  dofs_per_cell = fe.dofs_per_cell;
  qpts_per_cell = ipowf(n_q_points_1d,dim);

  // shape info, a single copy
  const internal::MatrixFreeFunctions::ShapeInfo<Number> shape_info(quad,fe);

  unsigned int size_shape_values = n_dofs_1d*n_q_points_1d*sizeof(Number);
  // test if  shape_info.shape_values_number.size() == (fe_degree+1)*num_quad_1d

  CUDA_CHECK_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(shape_values), &shape_info.shape_values_number[0],size_shape_values));

  if(update_flags & update_gradients) {
    CUDA_CHECK_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(shape_gradient), &shape_info.shape_gradient_number[0],size_shape_values));
  }

  // Setup number of cells per CUDA thread block
  cells_per_block = cells_per_block_shmem(dim,fe_degree);

  //---------------------------------------------------------------------------
  // cell-specific stuff (indices, JxW, inverse jacobian, quadrature points, etc)
  //---------------------------------------------------------------------------

  ReinitHelper<dim,Number> helper(this,mapping,fe,quad,shape_info,
                                  dof_handler,update_flags);

  if(use_coloring) {

    // create graph coloring
    typedef FilteredIterator<typename DoFHandler<dim>::active_cell_iterator> CellFilter;

    std::vector<std::vector<CellFilter > > graph =
      GraphColoringWrapper<dim>::make_graph_coloring(dof_handler,constraints);

    num_colors = graph.size();

    helper.setup_color_arrays(num_colors);

    for(int c = 0; c < num_colors; ++c) {
      n_cells[c] = graph[c].size();

      helper.setup_cell_arrays(c);

      unsigned int cellid=0;

      typename std::vector<CellFilter>::iterator
        cell = graph[c].begin(),
        end = graph[c].end();
      for(; cell != end; ++cell, ++cellid)
      {
        helper.get_cell_data(*cell,cellid);
      }


      // now allocate and copy stuff to the device

      helper.alloc_and_copy_arrays(c);

    }

  }
  else { // no coloring
    num_colors = 1;

    helper.setup_color_arrays(num_colors);

    n_cells[0] = n_cells_tot;

    helper.setup_cell_arrays(0);

    // loop over cells and extract data
    unsigned int cellid=0;
    typename DoFHandler<dim>::active_cell_iterator
      cell = dof_handler.begin_active(),
      endc = dof_handler.end();
    for (; cell!=endc; ++cell,++cellid)
    {
      helper.get_cell_data(cell,cellid);
    } // end cell loop

    // now allocate and copy stuff to the device
    helper.alloc_and_copy_arrays(0);

  }

  // setup row starts
  rowstart[0] = 0;
  for(int c = 1; c < num_colors; ++c) {
    rowstart[c] = rowstart[c-1] +  n_cells[c] * get_rowlength();
  }

  //---------------------------------------------------------------------------
  // constrained indices
  //---------------------------------------------------------------------------

  n_constrained_dofs = constraints.n_constraints();

  const unsigned int constr_num_blocks = ceil(n_constrained_dofs / float(MATRIX_FREE_BKSIZE_CONSTR));
  const unsigned int constr_x_num_blocks = round(sqrt(constr_num_blocks)); // get closest to even square.
  const unsigned int constr_y_num_blocks = ceil(double(constr_num_blocks)/constr_x_num_blocks);

  constr_grid_dim = dim3(constr_x_num_blocks,constr_y_num_blocks);
  constr_block_dim = dim3(MATRIX_FREE_BKSIZE_CONSTR);

  std::vector<unsigned int> constrained_dofs_host(n_constrained_dofs);

  unsigned int iconstr = 0;
  for(unsigned int i=0; i<n_dofs; i++) {
    if(constraints.is_constrained(i)) {
      constrained_dofs_host[iconstr] = i;
      iconstr++;
    }
  }

  CUDA_CHECK_SUCCESS(hipMalloc(&constrained_dofs,n_constrained_dofs*sizeof(unsigned int)));
  CUDA_CHECK_SUCCESS(hipMemcpy(constrained_dofs, &constrained_dofs_host[0],n_constrained_dofs*sizeof(unsigned int),
                                hipMemcpyHostToDevice));
}


template <int dim, typename Number>
void MatrixFreeGpu<dim,Number>::free()
{

  for(int c = 0; c < quadrature_points.size(); ++c) {
    if(quadrature_points[c] != NULL) CUDA_CHECK_SUCCESS(hipFree(quadrature_points[c]));
  }
  for(int c = 0; c < loc2glob.size(); ++c) {
    if(loc2glob[c] != NULL)          CUDA_CHECK_SUCCESS(hipFree(loc2glob[c]));
  }
  for(int c = 0; c < inv_jac.size(); ++c) {
    if(inv_jac[c] != NULL)           CUDA_CHECK_SUCCESS(hipFree(inv_jac[c]));
  }
  for(int c = 0; c < JxW.size(); ++c) {
    if(JxW[c] != NULL)               CUDA_CHECK_SUCCESS(hipFree(JxW[c]));
  }
  for(int c = 0; c < constraint_mask.size(); ++c) {
    if(constraint_mask[c] != NULL)   CUDA_CHECK_SUCCESS(hipFree(constraint_mask[c]));
  }

  quadrature_points.clear();
  loc2glob.clear();
  inv_jac.clear();
  JxW.clear();
  constraint_mask.clear();


  if(constrained_dofs != NULL)  CUDA_CHECK_SUCCESS(hipFree(constrained_dofs));
  constrained_dofs = NULL;
}


//=============================================================================
// functions dealing with constraints
//=============================================================================

template <typename Number>
__global__ void copy_constrained_dofs (Number              *dst,
                                       const Number        *src,
                                       const unsigned int  *constrained_dofs,
                                       const unsigned int  n_constrained_dofs)
{
  const unsigned int dof = threadIdx.x + blockDim.x*(blockIdx.x+gridDim.x*blockIdx.y);
  if(dof < n_constrained_dofs) {
    dst[constrained_dofs[dof]] = src[constrained_dofs[dof]];
  }
}

template <int dim, typename Number>
void MatrixFreeGpu<dim,Number>::copy_constrained_values(GpuVector <Number> &dst,
                                                        const GpuVector<Number> &src) const
{
  copy_constrained_dofs<Number> <<<constr_grid_dim,constr_block_dim>>>(dst.getData(),src.getDataRO(),
                                                                       constrained_dofs,
                                                                       n_constrained_dofs);
  CUDA_CHECK_LAST;
}


template <typename Number>
__global__ void set_constrained_dofs (Number               *dst,
                                      Number               val,
                                      const unsigned int   *constrained_dofs,
                                      const unsigned int   n_constrained_dofs)
{
  const unsigned int dof = threadIdx.x + blockDim.x*(blockIdx.x+gridDim.x*blockIdx.y);
  if(dof < n_constrained_dofs) {
    dst[constrained_dofs[dof]] = val;
  }
}


template <int dim, typename Number>
void MatrixFreeGpu<dim,Number>::set_constrained_values(GpuVector <Number> &dst,
                                                       Number val) const
{
  set_constrained_dofs<Number> <<<constr_grid_dim,constr_block_dim>>>(dst.getData(),
                                                                      val,constrained_dofs,
                                                                      n_constrained_dofs);
  CUDA_CHECK_LAST;
}
